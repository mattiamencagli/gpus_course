
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

//TODO add the necessary CUDA headers

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

//TODO copy the cpu version of the function and translate it into CUDA

void cpu_vec_add(double *a, double *b, double *c, const int &N) {
    for (int i = 0; i < N; ++i)
        c[i] = a[i] + b[i];
}


int main(int argc, char **argv){

    size_t N = 1e8;
    if (argc > 1)
        N = atoi(argv[1]);
    size_t size = N * sizeof(double);

    printf("Total required memory: %1.3f GBytes\n", 3 * size * 1e-9);

    //TODO allocate memory in host RAM using the CUDA function
    double *h_a, *h_b, *h_c;
    h_a = (double *)malloc(size);
    h_b = (double *)malloc(size);
    h_c = (double *)malloc(size);

    for (size_t i = 0; i < N; ++i){
        h_a[i] = 1.0;
        h_b[i] = 2.0;
    }

    // CUDA events to count the execution time
    hipEvent_t start, stop, start_c, stop_c;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventCreate(&start_c));
    CUDA_SAFE_CALL(hipEventCreate(&stop_c));

    // start to count execution time
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start));

    //TODO Allocate memory space on the device 

    //TODO copy NEEDED matrices from host to device memory

    CUDA_SAFE_CALL(hipEventRecord(start_c, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start_c));

    //TODO Use the CUDA function, remember to define blocks and threads.
    cpu_vec_add(h_a, h_b, h_c, N);    

    CUDA_SAFE_CALL(hipEventRecord(stop_c, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop_c));

    //TODO Transfer results from device to host 

    // time counting terminate
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // compute time elapse on GPU computing
    float gpu_elapsed_time_ms, gpu_comp_elapsed_time_ms;
    CUDA_SAFE_CALL(hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&gpu_comp_elapsed_time_ms, start_c, stop_c));
    printf("Time: %f ms.\n", gpu_elapsed_time_ms);
    printf("Time computation: %f ms.\n", gpu_comp_elapsed_time_ms);
    printf("print the first value of c (should be 3.0 if you do not change the values within the vectors) : \n %1.1f\n", h_c[0]);
    
    //TODO free memory (both host and device) with the CUDA function
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;

}
