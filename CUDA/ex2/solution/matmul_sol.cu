#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>


#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void gpu_matmul(double *a, double *b, double *c, int N) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x; // col
    int j = blockIdx.y * blockDim.y + threadIdx.y; // raw
    double sum = 0.0;
    if( i < N && j < N ) {
        for(int k = 0; k < N; k++)
            sum += a[k + j * N] * b[i + k * N];
        c[i + j * N] = sum;
    }
} 

void cpu_matmul(double *a, double *b, double *c, const int &N) {
    for (int j = 0; j < N; ++j)
        for (int i = 0; i < N; ++i) {
            double sum = 0.0;
            for (int k = 0; k < N; ++k)
                sum += a[k + j * N] * b[i + k * N];
            c[i + j * N] = sum;
        }
}


int main(int argc, char **argv){

    size_t N = 1e4;
    if (argc > 1)
        N = atoi(argv[1]);
    size_t N2 = N * N;
    size_t size = N2 * sizeof(double);

    printf("Total required memory: %1.3f GBytes\n", 3.0 * double(size) * 1e-9);

    // allocate memory in host RAM
    double *h_a, *h_b, *h_c;
    CUDA_SAFE_CALL(hipHostMalloc(&h_a, size, hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostMalloc(&h_b, size, hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostMalloc(&h_c, size, hipHostMallocDefault));

    for (size_t i = 0; i < N2; ++i){
        h_a[i] = 1.0;
        h_b[i] = 2.0;
    }

    // CUDA events to count the execution time
    hipEvent_t start, stop, start_c, stop_c;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventCreate(&start_c));
    CUDA_SAFE_CALL(hipEventCreate(&stop_c));

    // start to count execution time
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start));

    // Allocate memory space on the device 
    double *d_a, *d_b, *d_c;
    CUDA_SAFE_CALL(hipMalloc(&d_a, size));
    CUDA_SAFE_CALL(hipMalloc(&d_b, size));
    CUDA_SAFE_CALL(hipMalloc(&d_c, size));

    // copy matrix A and B from host to device memory
    CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipEventRecord(start_c, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start_c));

    dim3 threads(16, 16, 1);
    dim3 blocks((N - 1) / threads.x + 1, (N - 1) / threads.y + 1, 1);
    gpu_matmul<<<blocks, threads>>>(d_a, d_b, d_c, N);

    CUDA_SAFE_CALL(hipEventRecord(stop_c, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop_c));

    // Transefr results from device to host 
    CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    // time counting terminate
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // compute time elapse on GPU computing
    float gpu_elapsed_time_ms, gpu_comp_elapsed_time_ms;
    CUDA_SAFE_CALL(hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&gpu_comp_elapsed_time_ms, start_c, stop_c));
    printf("Time: %f ms.\n", gpu_elapsed_time_ms);
    printf("Time computation: %f ms.\n", gpu_comp_elapsed_time_ms);
    printf("print the first value of c (should be %d if you do not change the values within the vectors) : \n %1.1f\n", N*2 , h_c[0]);
    
    // free memory
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_b));
    CUDA_SAFE_CALL(hipFree(d_c));
    CUDA_SAFE_CALL(hipHostFree(h_a));
    CUDA_SAFE_CALL(hipHostFree(h_b));
    CUDA_SAFE_CALL(hipHostFree(h_c));

    return 0;

}
