
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <cstdint>
#include <string>

//TODO add the necessary CUDA headers

#define SOFTENING 1e-9f

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// Each body contains x, y, and z coordinate positions, as well as velocities in the x, y, and z directions.
typedef struct { float x, y, z, vx, vy, vz; } Body;

void read_values_from_file(const char * file, Body * data, size_t size) {
    std::ifstream values(file, std::ios::binary);
    if(values.fail()){
        printf("fail trying to open the file: %s\n", file);
        exit(1);
    }
    values.read(reinterpret_cast<char*>(data), size);
    values.close();
}

void write_values_to_file(const char * file, Body * data, size_t size) {
    std::ofstream values(file, std::ios::binary);
    if(values.fail()){
        printf("fail trying to open the file: %s\n", file);
        exit(1);
    }
    values.write(reinterpret_cast<char*>(data), size);
    values.close();
}

void check_correctness(const char * file_out, const char * file_sol, size_t size, size_t nBodies){

    Body *out = (Body *)malloc(size);
    Body *sol = (Body *)malloc(size);
    std::ifstream values_output(file_out, std::ios::binary);
    std::ifstream values_solution(file_sol, std::ios::binary);    
    if(values_output.fail()){
        printf("fail trying to open the file: %s\n", file_out);
        exit(1);
    } else if(values_output.fail()) {
        printf("fail trying to open the file: %s\n", file_sol);
        exit(1);
    }
    values_output.read(reinterpret_cast<char*>(out), size);
    values_solution.read(reinterpret_cast<char*>(sol), size);

    for(int i=0; i<nBodies; ++i)
        if(out[i].x != sol[i].x ){
            printf("\n\e[01;31m YOUR OUTPUT IS WRONG!\e[0;37m :(\n\n");
            printf("output body %d    : %f %f %f %f %f %f\n", i, out[i].x, out[i].y, out[i].z, out[i].vx, out[i].vy, out[i].vz);
            printf("solution body %d  : %f %f %f %f %f %f\n", i, sol[i].x, sol[i].y, sol[i].z, sol[i].vx, sol[i].vy, sol[i].vz);
            exit(1);
        }

    values_output.close();
    values_solution.close();

    free(out);
    free(sol);

    printf("\n\e[01;32m YOUR OUTPUT IS CORRECT!\e[0;37m :D\n\n");

}



//TODO copy the cpu version of the function and translate it into CUDA

void bodyForce(Body *p, float dt, int n) {
    for (int i = 0; i < n; ++i) {
        float Fx = 0.0f; 
        float Fy = 0.0f; 
        float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3; 
            Fy += dy * invDist3; 
            Fz += dz * invDist3;
        }

        p[i].vx += dt*Fx; 
        p[i].vy += dt*Fy; 
        p[i].vz += dt*Fz;
    }
}


//TODO copy the cpu version of the function and translate it into CUDA

void integratePosition(Body* p, float dt, int n) {

    for (int i = 0 ; i < n; i++) {
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

}


int main(int argc, char** argv) {

    // the operator << is the left bit shift operator. 
    // 1 in binary is still "1", if you shift it by 12 position you add 12 zeros obtaining: "100000000000", that is 2^12=4096.

    int nBodies = 1<<12; //the other choice is 1<<16
    if (argc > 1) 
        nBodies = 1<<atoi(argv[1]);
    int size = nBodies * sizeof(Body);

    const char *initialized_values, *output_values, *solution_values;

    if (nBodies == 1<<12) {
        initialized_values = "files/initialized_4096";
        output_values = "files/output_4096";
        solution_values = "files/solution_4096";
    } else if (nBodies == 1<<16) {
        initialized_values = "files/initialized_65536";
        output_values = "files/output_65536";
        solution_values = "files/solution_65536";
    } else {
        printf("ERROR: you must choose 12 or 16 for 4096 or 65536 bodies respectively!\n");
        exit(1);
    }

    hipEvent_t start, stop, start_i, stop_i, start_writing, stop_writing, start_reading, stop_reading;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventCreate(&start_i));
    CUDA_SAFE_CALL(hipEventCreate(&stop_i));
    CUDA_SAFE_CALL(hipEventCreate(&start_writing));
    CUDA_SAFE_CALL(hipEventCreate(&stop_writing));
    CUDA_SAFE_CALL(hipEventCreate(&start_reading));
    CUDA_SAFE_CALL(hipEventCreate(&stop_reading));


    //TODO get the device Id and th number of Streaminbg multiprocessors

    //TODO Allocate memory space on the device using the managed memory (or not, if you prefer to menage the memory by yourself)
    Body *bodies;
    bodies = (Body *)malloc(size);

    CUDA_SAFE_CALL(hipEventRecord(start_reading, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start_reading));   
    read_values_from_file(initialized_values, bodies, size);
    CUDA_SAFE_CALL(hipEventRecord(stop_reading, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop_reading));   

    //TODO Prefetch the memory to optimize data transfer

    //TODO Define blocks and threads.

    const float dt = 0.01f;  // Time step
    const int nIters = 10;  // Simulation iterations
    /*
    * This simulation will run for 10 cycles of time, calculating gravitational
    * interaction amongst bodies, and adjusting their positions according to their new velocities.
    */
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start));
    for (int iter = 0; iter < nIters; iter++) {

        CUDA_SAFE_CALL(hipEventRecord(start_i, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(start_i));

        //TODO Use the CUDA function
        bodyForce(bodies, dt, nBodies);
        
        //TODO remeber to wait for the GPU to finish before going to the next kernel.

        //TODO Use the CUDA function
        integratePosition(bodies, dt, nBodies);

        //TODO remeber to wait for the GPU to finish before going to the next kernel.
   
        CUDA_SAFE_CALL(hipEventRecord(stop_i, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop_i));

        float time_iter_ms;
        CUDA_SAFE_CALL(hipEventElapsedTime(&time_iter_ms, start_i, stop_i));
        printf("time iter %d iteration : %f ms\n", iter, time_iter_ms);

    }
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));   
    
    //TODO If you DO NOT use the managed memory, remeber to transfer the results back on the host
    CUDA_SAFE_CALL(hipEventRecord(start_writing, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(start_writing));   
    write_values_to_file(output_values, bodies, size);
    CUDA_SAFE_CALL(hipEventRecord(stop_writing, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop_writing));   
    
    float totalTime_loop_ms, time_writing_ms, time_reading_ms;
    CUDA_SAFE_CALL(hipEventElapsedTime(&totalTime_loop_ms, start, stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&time_writing_ms, start_writing, stop_writing));
    CUDA_SAFE_CALL(hipEventElapsedTime(&time_reading_ms, start_reading, stop_reading));

    printf("\ntime reading : %f ms\n", time_reading_ms);
    printf("time writing : %f ms\n\n", time_writing_ms);

    float avgTime_ms = totalTime_loop_ms / nIters;
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / (avgTime_ms * 1e-3);
    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);
    printf("TOT time loop : %f ms \n", totalTime_loop_ms);
    //TODO free memory (both host and device) with the CUDA function
    free(bodies);

    check_correctness(output_values, solution_values, size, nBodies);

    return 0;

}
